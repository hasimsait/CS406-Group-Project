
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <omp.h>
#include <math.h>       /* fabsf */
#include <string.h>
#include <stdlib.h>
#include <time.h>

#define DEBUG 0

//Error check-----
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}
//Error check-----
//This is a very good idea to wrap your calls with that function.. Otherwise you will not be able to see what is the error.
//Moreover, you may also want to look at how to use cuda-memcheck and cuda-gdb for debugging.

__global__ void parallel_cycles(int* d_xadj,int* d_adj, int* d_nv, int* d_result){
  
  //TO DO: GPU SCALE
  printf("Number of vertices: %d \n", *d_nv );
  
  
}

void wrapper(int* xadj, int* adj,int* no_vertices, int k){
  
  printf("Wrapper here! \n");


  int no_thread = 8;

  int* d_xadj;
  int* d_adj;
  int* d_result;
  int* h_result;
  int* d_nv;
 
  h_result = (int*)malloc(sizeof(int));
  *h_result = 0;



  //TO DO: DRIVER CODE
  hipSetDevice(0);

  hipEvent_t start, stop;
  float elapsedTime;

  hipMalloc( (void **) &d_xadj, *xadj *sizeof(int));
  hipMalloc( (void **) &d_adj, *adj *sizeof(int));
  hipMalloc( (void **) &d_result, sizeof(int));
  hipMalloc( (void **) &d_nv, sizeof(int));


  hipMemcpy(d_xadj, xadj, (*no_vertices) * sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy(d_adj, adj, (*no_vertices) * sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy(d_result, h_result, sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy(d_nv, no_vertices, sizeof(int), hipMemcpyHostToDevice );

  int no_blocks = (ceil)((*no_vertices)/no_thread);
  
  hipEventCreate(&start);
  hipEventRecord(start, 0);
  
  parallel_cycles<<<1,no_thread>>>(d_xadj, d_adj, d_nv, d_result);
  hipDeviceSynchronize(); 
  gpuErrchk( hipDeviceSynchronize() );
  
  
  hipEventCreate(&stop);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("GPU scale took: %f s\n", elapsedTime/1000);
  
    
}